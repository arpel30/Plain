#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "allFunctions.h"
#include <stdio.h>
#include <string.h>

// strings to lowercase
__device__ char tolowerCase(unsigned char ch) {
    if (ch >= 'A' && ch <= 'Z')
        ch = 'a' + (ch - 'A');
    return ch;
}

// compare strings not sensitive to case
__device__ int strcasecmpr(const char *s1, const char *s2) {
    const unsigned char *us1 = (const u_char *)s1,
                        *us2 = (const u_char *)s2;

    while (tolowerCase(*us1) == tolowerCase(*us2++))
        if (*us1++ == '\0')
            return (0);
    return (tolowerCase(*us1) - tolowerCase(*--us2));
}

__device__ int search_noExpand(char* valid, long size, char* word, int word_size)
{
	printf("word = %s\n", word);
	int offset=0, isSearch=1;
	for(int i=0; i<size; i++){
		if(valid[i] < '0' || valid[i] > '9')	
		{
			if(isSearch)
			{
				if(valid[i] == word[offset])
				{
					offset++;
					if(offset == word_size)
						return 1;
				}else
				{
					offset = 0;
					isSearch = 0;
				}
			}else
			{

			}
		}	
		else
		{
			isSearch = 1;
		}
	}
	return 0;
}

// check if the word is valid (only English letters)
__device__ int is_letters(char* word, int word_size)
{
	int i;
	for(i=0; i<word_size; i++)
	{
		if(!((word[i] >= 'a' && word[i] <= 'z') || (word[i] >= 'A' && word[i] <= 'Z')))
		{
			return 0;
		}
	}
	return 1;
}

// search word in 1d dictionary
__device__ int find_word(char* valid, long valid_size, char* word, int word_size)
{
	if(word_size <= 2) // statistically, there will be many errors in decoding that will result in words of 1-3 random letters
		return 0;
	if(!is_letters(word, word_size)) // search only if word contains only letters
		return 0;
	char* dict_word = (char*)malloc(20 * sizeof(char));
	if(!dict_word)
		return 0;
	int i=0, size=0, digits=1;
	while(i<valid_size)
	{
		// printf("find_word : %s, %d, %d\n", word, word_size, i);
		if(valid[i] >= '0' && valid[i] <= '9')
		{
			if(!strcasecmpr(word, dict_word)) // words are equal
			{
				free(dict_word);
				return 1;
			}
			size = size*(digits-1)*10 + valid[i] - '0';
			digits++;
			i++;
		}else if(valid[i] != 127) // DEL char
		{
			digits = 1;
			for(int j=0; j<size; j++)
			{
				dict_word[j] = valid[i];
				i++;
			}
			dict_word[size] = '\0';
		}else{
			i++;
		}
	}
	free(dict_word);
	return 0;
}

// search word in 2d dictionary
__device__ int search_word(char** dict, long size, char* word)
{
	for(int i=0; i<size; i++)
	{
		if(!strcasecmpr(word, dict[i]))
		{
			printf("%s = %s\n", word, dict[i]);
			return 1;
		}
	}
	return 0;
}

// expand 1d dictionary into 2d
__device__ void expand(char** dict, unsigned long valid_size, char* valid, int* longest_word)
{
	int size=0, line=0, digits=1;
	long i=0;
	*longest_word = 0;
	while(i<valid_size)
	{
		if(valid[i] >= '0' && valid[i] <= '9')
		{
			size = size*(digits-1)*10 + valid[i] - '0';
			digits++;
			i++;
			if(size > *longest_word)
				*longest_word = size;
		}else if(valid[i] != 127)
		{
			digits = 1;
			dict[line] = (char*)malloc((size+1) * sizeof(char));
			for(int j=0; j<size; j++)
			{
				dict[line][j] = valid[i];
				i++;
			}
			dict[line][size] = '\0';
			line++;
		}else{
			i++;
		}
	}
}

// check decrypted text in 2d dictionary
__device__ void is_english(int* ans, char** dict, long dict_size, char* plain, int size, int longest_word)
{
	int i=0, offset=0;
	char* word = (char*)malloc(longest_word * sizeof(char));
	while(i<size)
	{
		if(offset == longest_word)
			offset = 0;
		if(plain[i] == ' ')
		{
			word[offset] = '\0';
			*ans += search_word(dict, dict_size, word);
			offset = 0;
		}else{
			word[offset] = plain[i];
			offset++;
		}
		i++;
	}
	free(word);

}

// check decrypted text in 1d dictionary
__device__ void is_english_noExpand(int* ans, char* dict, long dict_size, char* plain, int size, int longest_word)
{
	int i=0, offset=0;
	char* word = (char*)malloc(longest_word * sizeof(char));
	if(!word)
		return;
	while(i<size)
	{
		if(offset == longest_word)
			offset = 0;
		if(plain[i] == ' ')
		{
			word[offset] = '\0';
			*ans += find_word(dict, dict_size, word, offset);
			offset = 0;
		}else{
			word[offset] = plain[i];
			offset++;
		}
		i++;
	}
	free(word);

}

// xor decryption with key
__device__ void xor_decrypt(char* cipher, unsigned long size, int key_size, long key_val, char* plain)
{
	char* key = (char*)malloc((key_size+1)*sizeof(char));
    for(int j=0;j<key_size;j++)
	{
        key[j]=(int)(key_val/powf(2,(j)*8))%(int)powf(2,(j+1)*8);
    }
		key[key_size] = '\0';
	char tmp;
	int i;
	for(i=0; i<size; i++)
	{
		tmp = cipher[i]^key[i%key_size];
		if((tmp == '\n' || tmp == '.' || tmp == '\r' || tmp == ','))
			plain[i] = ' ';
		else
			plain[i] = tmp; 
	}
	free(key);
}

__global__ void cuda_brute(char* cipher, unsigned long size, int key_size, char* valid, long valid_size, long num_rows, unsigned long partStart, unsigned long partStop) {

	unsigned long i = blockDim.x * blockIdx.x + threadIdx.x + partStart;
	int ans=0, longest_word=20;
	if(i < partStop+1) // just in key range
	{
		char* plain = (char*)malloc(size * sizeof(char));
		xor_decrypt(cipher, size, key_size, i, plain);
		if(num_rows <= 10)
		{
			char** dict = (char**)malloc(num_rows * sizeof(char*));
			expand(dict, valid_size, valid, &longest_word);
			is_english(&ans, dict, num_rows, plain, size, longest_word);
			for(int q=0; q<num_rows; q++)
			{
				free(dict[q]);
			}
			free(dict);
		}else{
			is_english_noExpand(&ans, valid, valid_size, plain, size, longest_word);
		}
		if(ans > 3)
			printf("answer = %d\n", ans);
		free(plain);
	}
}

int compute_on_gpu(char* cipher, long size, int key_size, char* valid, long valid_size, long num_rows, unsigned long partStart, unsigned long partStop) {
    	// Error code to check return values for CUDA calls
    	hipError_t err = hipSuccess;
		hipDeviceReset();
    	// Allocate memory on GPU to copy the data from the host
    	char *private_cipher;
		char *private_valid;
		
	err = hipMalloc((void **)&private_cipher, size);
    	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
        }
        
	err = hipMalloc((void**)&private_valid, sizeof(char*) * valid_size);
    	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device memory dictionary - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
        }

    	// Copy data from host to the GPU memory
    err = hipMemcpy(private_cipher, cipher, size, hipMemcpyHostToDevice);
    	if (err != hipSuccess) {
			fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
    	}

	err = hipMemcpy(private_valid, valid, strlen(valid), hipMemcpyHostToDevice);
    	if (err != hipSuccess) {
			fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
    	}


    	// Launch the Kernel
    	int threadsPerBlock = 256;
    	int blocksPerGrid =((partStop - partStart) + threadsPerBlock - 1) / threadsPerBlock;
    	cuda_brute<<<blocksPerGrid, threadsPerBlock>>>(private_cipher, size, key_size, private_valid, valid_size, num_rows, partStart, partStop);
    	err = hipGetLastError();
    	if (err != hipSuccess) {
			fprintf(stderr, "Failed to launch vectorAdd kernel -  %s\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
    	}

		
    	// Free allocated memory on GPU
    if (hipFree(private_cipher) != hipSuccess) {
		// fprintf(stderr, "Failed to free device data - cipher - %s\n", hipGetErrorString(err));
		//exit(EXIT_FAILURE);
		// return 0;
	}
		
	if (hipFree(private_valid) != hipSuccess) {
		// fprintf(stderr, "Failed to free device data - valid - %s\n", hipGetErrorString(err));
		//exit(EXIT_FAILURE);
		// return 0;
	}
    return 0;
}
